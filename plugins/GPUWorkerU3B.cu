#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "helpers.cuh"
#include "data.h"

using namespace corelib;
using namespace corelib::data;

__device__ static void reduceEarlyIdx(const int *i_S, int &beg, int end, int i_g) {
  if (end == beg) { beg = end; return;}
  if (i_S[end - 1] <= i_g) {
    beg = end;
    return;
  } else if (i_S[beg] > i_g) {
    return;
  }

  // binary search the first value >= i_g, at this point i_S has at least 2 elements
  int p = 0;
  while (beg < end) {
    p = (beg + end) / 2;
    if (i_S[p] > i_g) {
      end = p;
    } else {
      beg = p + 1;
    }
  }
}

struct TContext {
  int stackbeg0, stackbeg1, stackbeg2, stackbeg3;
  int stackend0, stackend1, stackend2, stackend3;
  int m0, m1, m2, m3, m4;

  int tl;

  unsigned char stackio;
  int level;
  int beg, end;
};


__device__ static int firstLarger(int *arr, int beg, int end, int v) {
  while (beg < end) {
    int mid = (beg + end) / 2;
    if (arr[mid] <= v) {
      beg = mid + 1;
    } else {
      end = mid;
    }
  }
  return end;
}

__device__ static void probeEnd(const int *i_S, int beg, int &end, int tl, const TemporalEdge *Eg) {
  while (beg < end) {
    int mid = (beg + end) / 2;
    if (Eg[i_S[mid]].t <= tl) {
      beg = mid + 1;
    } else {
      end = mid;
    }
  }
}

__device__ static void dumpContext(
  int stackbeg0, int stackbeg1, int stackbeg2, int stackbeg3,
  int stackend0, int stackend1, int stackend2, int stackend3,
  int m0, int m1, int m2, int m3, int m4,
  
  
  int tl,
  unsigned char stackio,
  int level,
  int beg, int end, const int *i_S,
  
  // graph
  const TemporalEdge *Eg,
  const int *inEdgesV,
  const int *outEdgesV,
  
  // mem
  TContext *offload,
  int *offtop,
  int *offload_width
) {
  int width = 0;
  probeEnd(i_S, beg, end, tl, Eg);
  width = max(width, end - beg);
  auto stackiob = stackio;
  switch (level) {
    case 4: 
      stackiob >>= 1;
      i_S = stackiob & 1 ? outEdgesV : inEdgesV;
      probeEnd(i_S, stackbeg3, stackend3, tl, Eg);
      width = max(width, stackend3 - stackbeg3);
    case 3: 
      stackiob >>= 1;
      i_S = stackiob & 1 ? outEdgesV : inEdgesV;
      probeEnd(i_S, stackbeg2, stackend2, tl, Eg);
      width = max(width, stackend2 - stackbeg2);
    case 2:
      stackiob >>= 1;
      i_S = stackiob & 1 ? outEdgesV : inEdgesV;
      probeEnd(i_S, stackbeg1, stackend1, tl, Eg);
      width = max(width, stackend1 - stackbeg1);
    case 1:
      stackiob >>= 1;
      i_S = stackiob & 1 ? outEdgesV : inEdgesV;
      probeEnd(i_S, stackbeg0, stackend0, tl, Eg);
      width = max(width, stackend0 - stackbeg0);
  }

  if (width == 0) return;

  int offpos = atomicAdd(offtop, 1);

  // each thread 
  offload[offpos].beg = beg;
  offload[offpos].end = end;
  offload[offpos].level = level;
  offload[offpos].stackio = stackio;
  offload[offpos].tl = tl;

  offload_width[offpos] = width;

  offload[offpos].stackbeg0 = stackbeg0;
  offload[offpos].stackbeg1 = stackbeg1;
  offload[offpos].stackbeg2 = stackbeg2;
  offload[offpos].stackbeg3 = stackbeg3;
  offload[offpos].stackend0 = stackend0;
  offload[offpos].stackend1 = stackend1;
  offload[offpos].stackend2 = stackend2;
  offload[offpos].stackend3 = stackend3;
  offload[offpos].m0 = m0;
  offload[offpos].m1 = m1;
  offload[offpos].m2 = m2;
  offload[offpos].m3 = m3;
  offload[offpos].m4 = m4;

}

__global__ static void MotifMatching_Expand(
  int work,
  // graph
  const TemporalEdge *Eg, int numeg,
  const int *inEdgesV, const int *inEdgesR,
  const int *outEdgesV, const int *outEdgesR,
  const int *nodeFeature,

  // motif
  const MotifEdgeInfoV1 *minfo, int numem,

  // runtime
  int *yeild,
  int *source,
  TContext *offload,
  int *offtop,
  int *offtopn,
  int *offload_width,

  int *chunk_offset,

  unsigned long long *gcount
) {
  auto laneid = threadIdx.x % 32;

  int tid;
  if (laneid == 0) tid = atomicAdd(source, 32);
  tid = __shfl_sync(0xffffffff, tid, 0);
  if (tid >= work) return;

  int stackbeg0 = 0, stackbeg1 = 0, stackbeg2 = 0, stackbeg3 = 0;
  int stackend0 = 0, stackend1 = 0, stackend2 = 0, stackend3 = 0;
  int m0 = 0, m1 = 0, m2 = 0, m3 = 0, m4 = 0;

  unsigned char stackio = 0;
  int tl;
  unsigned long long count = 0;
  int i_g = tid;
  int level = 0;
  int beg = 0, end = 0, base;
  MotifEdgeInfoV1 mi;
  const int *i_S = nullptr;
  clock_t timeup = 0;
  
  
  while (tid < work) { // valid block
    tid = (tid / 32) + chunk_offset[laneid];

    if (tid < work) {
      auto loc = firstLarger(offload_width, 0, *offtop, tid);
      int offset = loc ? tid - offload_width[loc - 1] : tid;

      // load context from loc
      level = offload[loc].level;
      tl = offload[loc].tl;
      stackio = offload[loc].stackio;

      beg = offload[loc].beg + offset;
      end = min(beg + 1, offload[loc].end);

      stackbeg0 = offload[loc].stackbeg0 + offset;
      stackbeg1 = offload[loc].stackbeg1 + offset;
      stackbeg2 = offload[loc].stackbeg2 + offset;
      stackbeg3 = offload[loc].stackbeg3 + offset;
      stackend0 = min(stackbeg0 + 1, offload[loc].stackend0);
      stackend1 = min(stackbeg1 + 1, offload[loc].stackend1);
      stackend2 = min(stackbeg2 + 1, offload[loc].stackend2);
      stackend3 = min(stackbeg3 + 1, offload[loc].stackend3);
      m0 = offload[loc].m0;
      m1 = offload[loc].m1;
      m2 = offload[loc].m2;
      m3 = offload[loc].m3;
      m4 = offload[loc].m4;


      mi = minfo[level];
      mi.io = stackio & 1;
      i_S = mi.io ? outEdgesV : inEdgesV;
      if (mi.constraintNode >= 0) mi.constraintNode = mi.io ? mi.baseNode : mi.constraintNode;
    } else {
      end = beg = 0;
    }

    int loopCnt = 0;
    while (true) {
      i_g = numeg;
      int node;
      for (; beg < end; beg++) {
        auto idx = i_S[beg];
        auto eg = Eg[idx];
        if (eg.t > tl) {
          end = beg;
          break;
        }
        node = mi.io ? eg.v : eg.u;
        bool checked = true;
        if (mi.constraintNode < 0) {
          switch (mi.mappedNodes) {
            case 5: checked = (m4 != node);
            case 4: checked = checked && (m3 != node);
            case 3: checked = checked && (m2 != node);
            case 2: checked = checked && (m1 != node) && (m0 != node);
          }
        } else {
          switch (mi.constraintNode) {
            case 4: checked = (m4 == node); break; 
            case 3: checked = (m3 == node); break; 
            case 2: checked = (m2 == node); break;
            case 1: checked = (m1 == node); break;
            case 0: checked = (m0 == node); break;
          }
        }
        // nodeConstraint(checked, node);
        if (checked) {
          i_g = idx; 
          beg++;
          break;
        }
      }

      bool alive = (level || (i_g < numeg));

      if (i_g < numeg) { 
        if (level == numem - 2) { 
          count++;
        } else {
          switch (mi.mappedNodes) {
            case 2: m2 = node;
            case 3: m3 = node; 
            case 4: m4 = node; 
          }
          switch (level) {
            case 0: stackbeg0 = beg; stackend0 = end;
            case 1: stackbeg1 = beg; stackend1 = end;
            case 2: stackbeg2 = beg; stackend2 = end;
            case 3: stackbeg3 = beg; stackend3 = end;
          }
          level++; 

          mi = minfo[level];
          stackio <<= 1;
          if (mi.io >= 0) {
            switch (mi.baseNode) { 
              case 0: base = m0; break;
              case 1: base = m1; break;
              case 2: base = m2; break;
              case 3: base = m3; break;
              case 4: base = m4;
            }
            beg = mi.arrR[base];
            end = mi.arrR[base + 1];
            i_S = mi.arrV;
          } else {
            int base0, base1;
            switch (mi.baseNode) { 
              case 0: base0 = m0; break;
              case 1: base0 = m1; break;
              case 2: base0 = m2; break;
              case 3: base0 = m3; break;
              case 4: base0 = m4;
            }
            switch (mi.constraintNode) { 
              case 0: base1 = m0; break;
              case 1: base1 = m1; break;
              case 2: base1 = m2; break;
              case 3: base1 = m3; break;
              case 4: base1 = m4;
            }
            int beg0 = inEdgesR[base0];
            int end0 = inEdgesR[base0 + 1];
            int beg1 = outEdgesR[base1];
            int end1 = outEdgesR[base1 + 1];
            if (end0 - beg0 < end1 - beg1) {
              mi.io = 0;
              beg = beg0;
              end = end0;
              i_S = inEdgesV;
            } else {
              mi.io = 1;
              mi.constraintNode = mi.baseNode;
              beg = beg1;
              end = end1;
              i_S = outEdgesV;
            }
          }
          stackio |= mi.io ? 1 : 0;

          reduceEarlyIdx(i_S, beg, end, i_g);
        }
      } else { 
        if (level) {
          level--;
          switch (level) {
            case 0: beg = stackbeg0; end = stackend0; break;
            case 1: beg = stackbeg1; end = stackend1; break;
            case 2: beg = stackbeg2; end = stackend2; break;
            case 3: beg = stackbeg3; end = stackend3; break;
          }
          stackio >>= 1;

          mi = minfo[level];
          mi.io = stackio & 1;
          i_S = mi.io ? outEdgesV : inEdgesV;
          if(mi.constraintNode >= 0) mi.constraintNode = mi.io ? mi.baseNode : mi.constraintNode;
        }
      }

      if (loopCnt % 1024 == 0 && *yeild) {
        if (laneid == 0) {
          timeup = clock() + 100000;
        } 
        timeup = __shfl_sync(0xffffffff, timeup, 0);
      }

      if (loopCnt % 64 == 0 && timeup && __shfl_sync(0xffffffff, clock(), 0) > timeup) {
        break;
      }

      if (__any_sync(0xffffffff, alive) == 0) break;

      if (laneid == 0) loopCnt++;
      loopCnt = __shfl_sync(0xffffffff, loopCnt, 0);
    }

    if (laneid == 0) tid = atomicAdd(source, 32);
    tid = __shfl_sync(0xffffffff, tid, 0);
  } // end of outer loop

  for (int offset = 16; offset > 0; offset /= 2)
    count += __shfl_down_sync(0xffffffff, count, offset);

  if (laneid == 0) atomicAdd(gcount, count);

  atomicAdd(yeild, 1);
  
  if ((level || (beg < end))) {
    dumpContext(
      stackbeg0, stackbeg1, stackbeg2, stackbeg3,
      stackend0, stackend1, stackend2, stackend3,
      m0, m1, m2, m3, m4,
      
  
      tl,
  
      stackio, level, beg, end, i_S,
  
      Eg,
      inEdgesV,
      outEdgesV,
  
      offload,
      offtopn,
      offload_width
    );
  }
}


__global__ static void MotifMatching_dispatch(
  int work, int delta,
  // graph
  const TemporalEdge *Eg, int numeg,
  const int *inEdgesV, const int *inEdgesR,
  const int *outEdgesV, const int *outEdgesR,
  const int *nodeFeature,

  // motif
  const MotifEdgeInfoV1 *minfo, int numem,

  // runtime
  int *yeild,
  int *source,
  TContext *offload,
  int *offtop,
  int *offtopn,
  int *offload_width,
  
  unsigned long long *gcount
) {

  auto laneid = threadIdx.x % 32;
  int tid;
  if (laneid == 0) tid = atomicAdd(source, 32);
  tid = __shfl_sync(0xffffffff, tid, 0);
  if (tid >= work) return;

  int stackbeg0 = 0, stackbeg1 = 0, stackbeg2 = 0, stackbeg3 = 0;
  int stackend0 = 0, stackend1 = 0, stackend2 = 0, stackend3 = 0;
  int m0 = 0, m1 = 0, m2 = 0, m3 = 0, m4 = 0;

  unsigned char stackio = 0;
  int tl;
  unsigned long long count = 0;
  int i_g = tid;
  int level = 0;
  int beg = 0, end = 0, base;
  MotifEdgeInfoV1 mi;
  const int *i_S = nullptr;
  clock_t timeup = 0;
  
  while (tid < work) { // valid block
    tid += laneid;

    bool fcheck = tid < work && (Eg[tid].u != Eg[tid].v);
    // nodeConstraint(fcheck, Eg[tid].u);
    // nodeConstraint(fcheck, Eg[tid].v);

    if (fcheck) {
      m0 = Eg[tid].u;
      m1 = Eg[tid].v;
      tl = Eg[tid].t + delta;
      i_g = tid;
      mi = minfo[level];
      stackio <<= 1;
      if (mi.io >= 0) {
        switch (mi.baseNode) { 
          case 0: base = m0; break;
          case 1: base = m1; break;
          case 2: base = m2; break;
          case 3: base = m3; break;
          case 4: base = m4;
        }
        beg = mi.arrR[base];
        end = mi.arrR[base + 1];
        i_S = mi.arrV;
      } else {
        int base0, base1;
        switch (mi.baseNode) { 
          case 0: base0 = m0; break;
          case 1: base0 = m1; break;
          case 2: base0 = m2; break;
          case 3: base0 = m3; break;
          case 4: base0 = m4;
        }
        switch (mi.constraintNode) { 
          case 0: base1 = m0; break;
          case 1: base1 = m1; break;
          case 2: base1 = m2; break;
          case 3: base1 = m3; break;
          case 4: base1 = m4;

        }
        int beg0 = inEdgesR[base0];
        int end0 = inEdgesR[base0 + 1];
        int beg1 = outEdgesR[base1];
        int end1 = outEdgesR[base1 + 1];
        if (end0 - beg0 < end1 - beg1) {
          mi.io = 0;
          beg = beg0;
          end = end0;
          i_S = inEdgesV;
        } else {
          mi.io = 1;
          mi.constraintNode = mi.baseNode;
          beg = beg1;
          end = end1;
          i_S = outEdgesV;
        }
      }
      stackio |= mi.io ? 1 : 0;

      reduceEarlyIdx(i_S, beg, end, i_g);
    } else {
      end = beg = 0;
    }


    int loopCnt = 0;
    while (true) {
      i_g = numeg;
      int node;
      for (; beg < end; beg++) {
        auto idx = i_S[beg];
        auto eg = Eg[idx];
        if (eg.t > tl) {
          end = beg;
          break;
        }
        node = mi.io ? eg.v : eg.u;
        bool checked = true;
        if (mi.constraintNode < 0) {
          switch (mi.mappedNodes) {
            case 5: checked = (m4 != node);
            case 4: checked = checked && (m3 != node);
            case 3: checked = checked && (m2 != node);
            case 2: checked = checked && (m1 != node) && (m0 != node);
          }
        } else {
          switch (mi.constraintNode) {
            case 4: checked = (m4 == node); break; 
            case 3: checked = (m3 == node); break; 
            case 2: checked = (m2 == node); break;
            case 1: checked = (m1 == node); break;
            case 0: checked = (m0 == node); break;
          }
        }
        // nodeConstraint(checked, node);
        if (checked) {
          i_g = idx; 
          beg++;
          break;
        }
      }

      bool alive = (level || (i_g < numeg));

      if (i_g < numeg) { 
        if (level == numem - 2) { 
          count++;
        } else {
          switch (mi.mappedNodes) {
            case 2: m2 = node;
            case 3: m3 = node; 
            case 4: m4 = node; 
          }
          switch (level) {
            case 0: stackbeg0 = beg; stackend0 = end;
            case 1: stackbeg1 = beg; stackend1 = end;
            case 2: stackbeg2 = beg; stackend2 = end;
            case 3: stackbeg3 = beg; stackend3 = end;
          }

          level++; 

          mi = minfo[level];
          stackio <<= 1;
          if (mi.io >= 0) {
            switch (mi.baseNode) { 
              case 0: base = m0; break;
              case 1: base = m1; break;
              case 2: base = m2; break;
              case 3: base = m3; break;
              case 4: base = m4;
            }
            beg = mi.arrR[base];
            end = mi.arrR[base + 1];
            i_S = mi.arrV;
          } else {
            int base0, base1;
            switch (mi.baseNode) { 
              case 0: base0 = m0; break;
              case 1: base0 = m1; break;
              case 2: base0 = m2; break;
              case 3: base0 = m3; break;
              case 4: base0 = m4;
            }
            switch (mi.constraintNode) { 
              case 0: base1 = m0; break;
              case 1: base1 = m1; break;
              case 2: base1 = m2; break;
              case 3: base1 = m3; break;
              case 4: base1 = m4;
            }
            int beg0 = inEdgesR[base0];
            int end0 = inEdgesR[base0 + 1];
            int beg1 = outEdgesR[base1];
            int end1 = outEdgesR[base1 + 1];
            if (end0 - beg0 < end1 - beg1) {
              mi.io = 0;
              beg = beg0;
              end = end0;
              i_S = inEdgesV;
            } else {
              mi.io = 1;
              mi.constraintNode = mi.baseNode;
              beg = beg1;
              end = end1;
              i_S = outEdgesV;
            }
          }
          stackio |= mi.io ? 1 : 0;

          reduceEarlyIdx(i_S, beg, end, i_g);
        }
      } else { 
        if (level) {
          level--;
          switch (level) {
            case 0: beg = stackbeg0; end = stackend0; break;
            case 1: beg = stackbeg1; end = stackend1; break;
            case 2: beg = stackbeg2; end = stackend2; break;
            case 3: beg = stackbeg3; end = stackend3; break;
          }

          stackio >>= 1;

          mi = minfo[level];
          mi.io = stackio & 1;
          i_S = mi.io ? outEdgesV : inEdgesV;
          if(mi.constraintNode >= 0) mi.constraintNode = mi.io ? mi.baseNode : mi.constraintNode;
        }

      }

      if (loopCnt % 1024 == 0 && *yeild) {
        if (laneid == 0) {
          timeup = clock() + 100000;
        } 
        timeup = __shfl_sync(0xffffffff, timeup, 0);
      }

      if (loopCnt % 64 == 0 && timeup && __shfl_sync(0xffffffff, clock(), 0) > timeup) {
        break;
      }
        
      if (__any_sync(0xffffffff, alive) == 0) break;

      if (laneid == 0) loopCnt++;
      loopCnt = __shfl_sync(0xffffffff, loopCnt, 0);
    }

    if (laneid == 0) tid = atomicAdd(source, 32);
    tid = __shfl_sync(0xffffffff, tid, 0);
  } // end of outer loop

  for (int offset = 16; offset > 0; offset /= 2)
    count += __shfl_down_sync(0xffffffff, count, offset);

  if (laneid == 0) atomicAdd(gcount, count);

  atomicAdd(yeild, 1); 
  
  if ((level || (beg < end))) {
    dumpContext(
      stackbeg0, stackbeg1, stackbeg2, stackbeg3,
      stackend0, stackend1, stackend2, stackend3,
      m0, m1, m2, m3, m4,
      
  
      tl,
  
      stackio, level, beg, end, i_S,
  
      Eg,
      inEdgesV,
      outEdgesV,
  
      offload,
      offtop,
      offload_width
    );
  }
}

// --------------------------------- 

static unsigned long long TMotifMatchingGPUImpl(
                          int numBlocksA, int numBlocksB, int sizeBlock,
                          int work, int delta,
                          // graph
                          const TemporalEdge *Eg, int numeg,
                          const int *inEdgesV, const int *inEdgesR, const int *outEdgesV, const int *outEdgesR,
                          const int *nodeFeature,
                          // motif
                          const MotifEdgeInfoV1 *minfo, int numem,
                        
                          // runtime mem
                          int *yeild,
                          int *source,
                          TContext *offload,
                          int *offtop,
                          int *offtopn,
                          int *offload_width,

                          unsigned long long *gcount 
                          ) {
  hipError_t err;
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1); hipStreamCreate(&stream2);
  // inspector_u3b <<< 1, 32, 0, stream2 >>> (work);
  MotifMatching_dispatch <<< numBlocksA, sizeBlock, 0, stream1 >>>(
      work, delta,
      Eg, numeg,
      inEdgesV, inEdgesR, outEdgesV, outEdgesR,
      nodeFeature,

      minfo, numem,

      yeild,
      source,
      offload,
      offtop,
      offtopn,
      offload_width,

      gcount
  );
  err = hipGetLastError();

  if ( err != hipSuccess ) {
      printf("CUDA Error: %s\n", hipGetErrorString(err));       
      exit(-1);
  }

  int chunk_offset_h[32];
  int *chunk_offset;
  gpuErrchk(hipMalloc(&chunk_offset, sizeof(int) * 32));
  gpuErrchk(hipMemset(chunk_offset, 0, sizeof(int) * 32));

  int offload_cnt = 0;
  gpuErrchk(hipMemcpy(&offload_cnt, offtop, sizeof(int), hipMemcpyDeviceToHost));

  while (offload_cnt > 0) {
    int t = 0;
    gpuErrchk(hipMemcpy(source, &t, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(yeild, &t, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(offtopn, &t, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(offtop, &offload_cnt, sizeof(int), hipMemcpyHostToDevice));
    

    // prefix sum
    thrust::device_ptr<int> dev_ptr(offload_width);

    auto sump = thrust::inclusive_scan(dev_ptr, dev_ptr + offload_cnt, dev_ptr);
    auto sum = *(sump - 1);
    for (int i = 0; i < 32; i++) {
      chunk_offset_h[i] = i * (sum / 32 + (sum % 32 != 0));
    }
    gpuErrchk(hipMemcpy(chunk_offset, &chunk_offset_h, sizeof(int) * 32, hipMemcpyHostToDevice));

    MotifMatching_Expand <<< numBlocksB, sizeBlock >>> (sum,
      Eg, numeg,
      inEdgesV, inEdgesR, outEdgesV, outEdgesR,
      nodeFeature,

      minfo, numem,

      yeild,
      source,
      offload,
      offtop,
      offtopn,
      offload_width,

      chunk_offset,

      gcount
    );

    err = hipGetLastError();
    
    if ( err != hipSuccess ) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    gpuErrchk(hipMemcpy(&offload_cnt, offtopn, sizeof(int), hipMemcpyDeviceToHost));
  }


  hipDeviceSynchronize();  

  gpuErrchk(hipFree(chunk_offset));

  unsigned long long count_h;
  gpuErrchk(hipMemcpy(&count_h, gcount, sizeof(unsigned long long), hipMemcpyDeviceToHost));
  return count_h;
}

struct GPUWorkerU3B : public GPUWorker {
  // Execution Mem
  unsigned long long *count_d;

  int *yeild_d;
  int *source_d;
  TContext *offload_d;
  int *offtop, *offtopn;
  int *offload_width_d;

  GPUWorkerU3B(int gpu, int sizeBlock = 96);
  unsigned long long run() override;
  virtual void take(MineJob &job) override;
  void update_job() override;
  ~GPUWorkerU3B() override;
};

GPUWorkerU3B::GPUWorkerU3B(int gpu, int sizeBlock) : GPUWorker("U3B", gpu, sizeBlock) {
  gpuErrchk(hipSetDevice(gpu_));
  gpuErrchk(hipMalloc(&count_d, sizeof(unsigned long long)));
  gpuErrchk(hipMemset(count_d, 0, sizeof(unsigned long long)));

  gpuErrchk(hipMalloc(&yeild_d, sizeof(int)));
  gpuErrchk(hipMemset(yeild_d, 0, sizeof(int)));
  gpuErrchk(hipMalloc(&source_d, sizeof(int)));
  gpuErrchk(hipMemset(source_d, 0, sizeof(int)));
  gpuErrchk(hipMalloc(&offload_d, 2092 * 96 * sizeof(TContext)));
  gpuErrchk(hipMemset(offload_d, 0, 2092 * 96 * sizeof(TContext)));
  gpuErrchk(hipMalloc(&offtop, sizeof(int)));
  gpuErrchk(hipMalloc(&offtopn, sizeof(int)));
  gpuErrchk(hipMemset(offtop, 0, sizeof(int)));
  gpuErrchk(hipMemset(offtopn, 0, sizeof(int)));
  gpuErrchk(hipMalloc(&offload_width_d, 2092 * 96 * sizeof(int)));
  gpuErrchk(hipMemset(offload_width_d, 0, 2092 * 96 * sizeof(int)));
}

GPUWorkerU3B::~GPUWorkerU3B() {
  gpuErrchk(hipSetDevice(gpu_));
  gpuErrchk(hipFree(count_d));
  gpuErrchk(hipFree(yeild_d));
  gpuErrchk(hipFree(source_d));
  gpuErrchk(hipFree(offload_d));
  gpuErrchk(hipFree(offtop));
  gpuErrchk(hipFree(offtopn));
  gpuErrchk(hipFree(offload_width_d));
}

unsigned long long GPUWorkerU3B::run() {
  gpuErrchk(hipSetDevice(gpu_));
  // Get device properties
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, gpu_);

  int maxBlocksPerSM;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocksPerSM, MotifMatching_dispatch, sizeBlock_, 0);
  auto numBlocksA = maxBlocksPerSM * deviceProp.multiProcessorCount;

  hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocksPerSM, MotifMatching_Expand, sizeBlock_, 0);
  auto numBlocksB = maxBlocksPerSM * deviceProp.multiProcessorCount;

  auto &dd = job_->data;

  count_ = TMotifMatchingGPUImpl(
      numBlocksA, numBlocksB, sizeBlock_, 
      job_->end, job_->delta,

      dd->Eg_d, dd->graphNumEdges,
      dd->inEdgesV_d, dd->inEdgesR_d, dd->outEdgesV_d, dd->outEdgesR_d,
      dd->nodefeatures_d,

      dd->minfo(), dd->motifNumEdges,

      yeild_d,
      source_d,
      offload_d,
      offtop,
      offtopn,
      offload_width_d,

      count_d
  );
  return count_;
}


void GPUWorkerU3B::update_job() {
  job_->beg = job_->end;
}

void GPUWorkerU3B::take(MineJob &job) {
  gpuErrchk(hipSetDevice(gpu_));
  GPUWorker::take(job);

  gpuErrchk(hipMemset(yeild_d, 0, sizeof(int)));
  gpuErrchk(hipMemset(offtop, 0, sizeof(int)));
  gpuErrchk(hipMemset(offtopn, 0, sizeof(int)));
  gpuErrchk(hipMemcpy(source_d, &(job_->beg), sizeof(int), hipMemcpyHostToDevice));
}

extern "C" {
  GPUWorker *getWorker(int gpu);
}

GPUWorker *getWorker(int gpu) {
  return new GPUWorkerU3B(gpu);
}