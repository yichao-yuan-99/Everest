#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "helpers.cuh"

#include "data.h"

using namespace corelib;
using namespace corelib::data;

/*
 * u0 version implement search result memorization.
 */

// ---------- GPU kernels ---------- 
// return the first position that is >= i_g
__device__ static int binarySearchGeq(const int *i_S, int beg, int end, int i_g) {
    int p = 0, ans = 0;
    while (beg < end) {
        p = (beg + end) / 2;
        if (i_S[p] >= i_g) {
            end = p;
        } else {
            ans = p + 1;
            beg = p + 1;
        }
    }
    return ans;
}


__device__ static void reduceEarlyIdx(const int *i_S, int &beg, int end, int i_g) {
    if (i_S[end - 1] < i_g) {
        beg = end;
        return;
    } else if (i_S[beg] >= i_g) {
        return;
    }

    // binary search the first value >= i_g, at this point i_S has at least 2 elements
    beg = binarySearchGeq(i_S, beg, end, i_g);
}

__device__ void TMotifMatching_genBegEnd(const TemporalEdge *Eg, int numeg, int i_g,
                                    const TemporalEdge *Em, int numnm, int i_m,
                                    int *MapMg,
                                    const int *inEdgesV, const int *inEdgesR, 
                                    const int *outEdgesV, const int *outEdgesR,
                                    int tl, int &beg, int &end, const int *&i_S
                                    ) {
    auto em = Em[i_m];
    int umg = MapMg[em.u], vmg = MapMg[em.v];

    if (umg >= 0 && vmg >= 0) {
        int ibeg = inEdgesR[vmg], iend = inEdgesR[vmg + 1];
        int obeg = outEdgesR[umg], oend = outEdgesR[umg + 1];
        if (iend - ibeg < oend - obeg) {
            i_S = inEdgesV;
            beg = ibeg;
            end = iend;
        } else {
            i_S = outEdgesV;
            beg = obeg;
            end = oend;
        }
    } else if (umg >= 0) {
        int obeg = outEdgesR[umg], oend = outEdgesR[umg + 1];
        i_S = outEdgesV;
        beg = obeg;
        end = oend;
    } else if (vmg >= 0) {
        int ibeg = inEdgesR[vmg], iend = inEdgesR[vmg + 1];
        i_S = inEdgesV;
        beg = ibeg;
        end = iend;
    }

    if (end == beg) return;
    if (i_S) reduceEarlyIdx(i_S, beg, end, i_g);
}

__global__ static void MotifMatching_dispatch( int work, int delta,
  // graph
  const TemporalEdge *Eg, int numeg,
  const int *inEdgesV, const int *inEdgesR,
  const int *outEdgesV, const int *outEdgesR,
  const int *nodeFeature,

  // motif
  const TemporalEdge *Em, int numem, int numnm,

  // runtime
  unsigned long long *gcount
  ) {
  // ** SEC. 1, First Edge & Initialization
  auto tid = blockDim.x * blockIdx.x + threadIdx.x;
  auto laneid = threadIdx.x % 32;
  unsigned long long count = 0;

  int MapMg[5], edgeCount[5], Estackbeg[5], Estackend[5];
  const int *Estackis[4];
  MapMg[0] = -1;
  MapMg[1] = -1;
  MapMg[2] = -1;
  MapMg[3] = -1;
  MapMg[4] = -1;

  edgeCount[0] = 0;
  edgeCount[1] = 0;
  edgeCount[2] = 0;
  edgeCount[3] = 0;
  edgeCount[4] = 0;

  int stp = 0;
  int i_g = tid + 1;
  int i_m = 1;

  auto em = Em[i_m];
  const int *i_S = nullptr; // nullptr implies from i_g...numeg
  int beg = i_g, end = numeg;
  int tl;

  // check if loop is perserved
  bool perserve_loop = (Em[0].u == Em[0].v && Eg[tid].u == Eg[tid].v) || 
              (Em[0].u != Em[0].v && Eg[tid].u != Eg[tid].v);
  if (!perserve_loop) goto EXITKERNEL;
  if (tid >= work) goto EXITKERNEL;

  // map the first edge; 
  MapMg[Em[0].u] = Eg[tid].u;
  MapMg[Em[0].v] = Eg[tid].v;
  edgeCount[Em[0].u]++;
  edgeCount[Em[0].v]++;
  // set the time limit
  tl = Eg[tid].t + delta;


  // ** SEC. 2, Tree exploration
  TMotifMatching_genBegEnd(Eg, numeg, i_g, Em, numnm, i_m, MapMg, inEdgesV, inEdgesR, outEdgesV, outEdgesR, tl, beg, end, i_S);

  while (true) {
    int umg = MapMg[em.u], vmg = MapMg[em.v];
    i_g = numeg;
    for (; beg < end; beg++) {
      auto idx = i_S ? i_S[beg] : beg;
      auto eg = Eg[idx];
      if (eg.t > tl) {
        end = beg;
        break;
      }
      bool perserve_loop = (em.u == em.v && eg.u == eg.v) || (em.u != em.v && eg.u != eg.v);
      bool uIsMapped = false, vIsMapped = false;
      for (int i = 0; i < numnm; i++) {
        if (MapMg[i] == eg.u) uIsMapped = true;
        if (MapMg[i] == eg.v) vIsMapped = true;
      }
      bool canMapu = eg.u == umg || (umg == -1 && !uIsMapped);
      bool canMapv = eg.v == vmg || (vmg == -1 && !vIsMapped);
      if (perserve_loop && canMapu && canMapv) {
        i_g = idx; 
        beg++;
        break;
      }
    }

    if (i_g < numeg) { 
      auto eg = Eg[i_g];
      if (i_m == numem - 1) { // last edge
        count++;
      } else {
        MapMg[em.u] = eg.u;
        MapMg[em.v] = eg.v;
        Estackbeg[stp] = beg;
        Estackend[stp] = end;
        Estackis[stp] = i_S;
        stp++;
        edgeCount[em.u]++;  // mark the edge inuse;
        edgeCount[em.v]++;

        i_m++; // step forward an edge in M, update em
        em = Em[i_m];
        i_g++; // finish this edge
        TMotifMatching_genBegEnd(Eg, numeg, i_g, Em, numnm, i_m, MapMg, inEdgesV, inEdgesR, outEdgesV, outEdgesR, tl, beg, end, i_S);
      }
    } else { // Trace back if current edge is "too old"
      if (stp) {
        stp--;
        beg = Estackbeg[stp]; 
        end = Estackend[stp]; 
        i_S = Estackis[stp];
        i_m--;
        em = Em[i_m];
        if (--edgeCount[em.u] == 0) {
          MapMg[em.u] = -1;
        }
        if (--edgeCount[em.v] == 0) {
          MapMg[em.v] = -1;
        }
      } else {
        goto EXITKERNEL;
      }
    }
  }

  // SEC. 3 Output
EXITKERNEL:
  for (int offset = 16; offset > 0; offset /= 2)
    count += __shfl_down_sync(0xffffffff, count, offset);

  if (laneid == 0) atomicAdd(gcount, count);
}

// --------------------------------- 

static unsigned long long TMotifMatchingGPUImpl( int numBlocks, int sizeBlock, int work, int delta,
  // graph
  const TemporalEdge *Eg, int numeg,
  const int *inEdgesV, const int *inEdgesR, const int *outEdgesV, const int *outEdgesR,
  const int *nodeFeature,
  // motif
  const TemporalEdge *Em, int numem, int numnm,

  // runtime mem
  unsigned long long *gcount
                          ) {
    // start the GPU kernel
    MotifMatching_dispatch <<< numBlocks, sizeBlock >>>(
      work, delta,
      Eg, numeg,
      inEdgesV, inEdgesR, outEdgesV, outEdgesR,
      nodeFeature,

      Em, numem, numnm,

      gcount
    );
    hipError_t err = hipGetLastError();

    if ( err != hipSuccess ) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       
        exit(-1);
    }

    // get the result back
    unsigned long long count_h;
    gpuErrchk(hipMemcpy(&count_h, gcount, sizeof(unsigned long long), hipMemcpyDeviceToHost));
    return count_h;

}

struct GPUWorkerU0 : public GPUWorker {
  // Execution Mem
  unsigned long long *count_d;

  GPUWorkerU0(int gpu, int sizeBlock = 96);
  unsigned long long run() override;
  void update_job() override;
  ~GPUWorkerU0() override;
};

GPUWorkerU0::GPUWorkerU0(int gpu, int sizeBlock) : GPUWorker("U0", gpu, sizeBlock) {
  gpuErrchk(hipSetDevice(gpu_));
  gpuErrchk(hipMalloc(&count_d, sizeof(unsigned long long)));
  gpuErrchk(hipMemset(count_d, 0, sizeof(unsigned long long)));
}

GPUWorkerU0::~GPUWorkerU0() {
  gpuErrchk(hipSetDevice(gpu_));
  gpuErrchk(hipFree(count_d));
}

unsigned long long GPUWorkerU0::run() {
  gpuErrchk(hipSetDevice(gpu_));
  if (!job_) {
    throw std::runtime_error("run a GPU worker without a job");
  }

  gpuErrchk(hipSetDevice(this->gpu_));

  auto &dd = job_->data;

  count_ = TMotifMatchingGPUImpl(
      numBlocks(), sizeBlock_, 
      job_->end, job_->delta,

      dd->Eg_d, dd->graphNumEdges,
      dd->inEdgesV_d, dd->inEdgesR_d, dd->outEdgesV_d, dd->outEdgesR_d,
      dd->nodefeatures_d,

      dd->Em_d(), dd->motifNumEdges, dd->motifNumVertices,

      count_d
  );
  return count_;
}

void GPUWorkerU0::update_job() {
  job_->beg = job_->end;
}

extern "C" {
  GPUWorker *getWorker(int gpu);
}

GPUWorker *getWorker(int gpu) {
  return new GPUWorkerU0(gpu);
}